#include "hip/hip_runtime.h"
#include <semaphore.h>
#include <iostream>
using std::cout;
using std::endl;
#include <stdexcept>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h> 

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/gather.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>

#include "s6GPU.h"
#include "stopwatch.hpp"

#include "hashpipe.h"
#include <time.h>

//#define USE_TIMER
//#define USE_TOTAL_GPU_TIMER
//#define USE_MEM_TIMER
//#define USE_SEM_TIMER
#ifdef USE_TIMER
    bool use_timer=true;
#else
    bool use_timer=false;
#endif
#ifdef USE_TOTAL_GPU_TIMER
    bool use_total_gpu_timer=true;
#else
    bool use_total_gpu_timer=false;
#endif
#ifdef USE_MEM_TIMER
    bool use_mem_timer=true;
#else
    bool use_mem_timer=false;
#endif
#ifdef USE_SEM_TIMER
    bool use_sem_timer=true;
#else
    bool use_sem_timer=false;
#endif
float sum_of_times;
float sum_of_mem_times;

//#define TRACK_GPU_MEMORY
#ifdef TRACK_GPU_MEMORY
    bool track_gpu_memory=true;
#else
    bool track_gpu_memory=false;
#endif

bool use_thread_sync=false;

cufft_config_t cufft_config;

device_vectors_t * init_device_vectors() {

    device_vectors_t * dv_p  = new device_vectors_t;

	dv_p->raw_timeseries_p=0;
	dv_p->fft_data_p=0;          
	dv_p->fft_data_out_p=0;          
	dv_p->powspec_p=0;          
    dv_p->baseline_p=0;         
    dv_p->normalised_p=0;       
    dv_p->scanned_p=0;       
    dv_p->hit_baselines_p=0;  
    dv_p->hit_indices_p=0;  
    dv_p->hit_powers_p=0; 

#ifndef SOURCE_FAST
    dv_p->hit_indices_p      = new thrust::device_vector<int>();
    dv_p->hit_powers_p       = new thrust::device_vector<float>;
    dv_p->hit_baselines_p    = new thrust::device_vector<float>;
#endif

    return dv_p;
}

int init_device(int gpu_dev) {

#define PRINT_DEVICE_PROPERTIES
#ifdef PRINT_DEVICE_PROPERTIES
  	int nDevices;
  	hipGetDeviceCount(&nDevices);
	fprintf(stderr, "\nGPUs on this system:\n");
  	for (int i = 0; i < nDevices; i++) {
    	hipDeviceProp_t prop;
    	hipGetDeviceProperties(&prop, i);
    	fprintf(stderr, "Device Number: %d\n", i);
    	fprintf(stderr, "  Device name: %s\n", prop.name);
    	fprintf(stderr, "  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    	fprintf(stderr, "  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    	fprintf(stderr, "  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  	}
#endif

    int rv = hipSetDevice(gpu_dev);

    // TODO error checking
    return rv;
}

void delete_device_vectors( device_vectors_t * dv_p) {
// TODO - is the right way to deallocate thrust vectors?
    delete(dv_p->hit_indices_p);      
    delete(dv_p->hit_powers_p);       
    delete(dv_p->hit_baselines_p);    

    delete(dv_p);
}

void gpu_fini() {
    hipProfilerStop();  
}

inline void timer_start(Stopwatch & timer) {
	timer.start();
}

inline float timer_stop(Stopwatch & timer, const char * label) {
	timer.stop();
 	float elapsed_time = timer.getTime();
	cout << label << ":\t" << elapsed_time << endl;
	timer.reset(); 
	return elapsed_time;  
}

void create_fft_plan_1d(hipfftHandle* plan,
                            int          istride,
                            int          idist,
                            int          ostride,
                            int          odist,
                            int          nfft_,
                            size_t       nbatch,
							hipfftType    fft_type) {

	if(track_gpu_memory) get_gpu_mem_info("on entry to create_fft_plan_1d()");

    int rank      = 1;
    int nfft[]    = {nfft_};
    int inembed[] = {nfft[0]};
    //int idist     = inembed[0];
    int onembed[] = {nfft[0]};
    //int odist     = onembed[0];
    hipfftResult fft_ret = hipfftPlanMany(plan,
                                        rank, nfft,
                                        inembed, istride, idist,
                                        onembed, ostride, odist,
                                        fft_type, nbatch);
    if( fft_ret != HIPFFT_SUCCESS ) {
        throw std::runtime_error("hipfftPlanMany failed");
    }

	if(track_gpu_memory) get_gpu_mem_info("on exit from create_fft_plan_1d()");
}

inline void get_gpu_mem_info(const char * comment) {
    int rv;
    size_t free, total;
    double free_gb, total_gb, allocated_gb;
    rv = hipMemGetInfo(&free, &total);
    if(rv) {
        fprintf(stderr, "Error from hipMemGetInfo() : %d : %s\n", rv, hipGetErrorString(hipGetLastError()));
    } else {
        total_gb = (double)total/(1024*1024*1024);
        free_gb =  (double)free/(1024*1024*1024);
        allocated_gb = total_gb - free_gb;
        fprintf(stdout, "GPU memory total : %2.2lf GB    allocated : %2.2lf GB (%2.2f%%)    free : %2.2lf GB    (%s)\n", 
                total_gb, allocated_gb, (allocated_gb/total_gb)*100, free_gb, comment);
    }
} 

inline void print_current_time(const char * comment) {
    long            ms; // Milliseconds
    time_t          s;  // Seconds
    struct timespec spec;

    clock_gettime(CLOCK_REALTIME, &spec);

    s  = spec.tv_sec;
    ms = round(spec.tv_nsec / 1.0e6); // Convert nanoseconds to milliseconds
    if (ms > 999) {
        s++;
        ms = 0;
    }

    fprintf(stderr, "%s : %ld.%03ld unix time\n", comment, s, ms);
}

// Note: input == output is ok
void execute_fft_plan_c2c(hipfftHandle   *plan,
                          const float2* input,
                          float2*       output) {
    hipfftResult fft_ret = hipfftExecC2C(*plan,
                                       (hipfftComplex*)input,
                                       (hipfftComplex*)output,
                                       HIPFFT_FORWARD);
    if( fft_ret != HIPFFT_SUCCESS ) {
        throw std::runtime_error("hipfftExecC2C failed");
    }
}

// Note: input == output is not ok
void execute_fft_plan_r2c(hipfftHandle   *plan,
                          const float*  input,
                          float2*       output) {
	hipfftResult fft_ret = hipfftExecR2C(*plan, 
									   (hipfftReal*) input, 
									   (hipfftComplex*) output);
    if( fft_ret != HIPFFT_SUCCESS ) {
        throw std::runtime_error("hipfftExecR2C failed");
    }
}

// Functors
// --------
struct convert_complex_8b_to_float
    : public thrust::unary_function<char2,float2> {
    inline __host__ __device__
    float2 operator()(char2 a) const {
        return make_float2(a.x, a.y);
    }
};
struct convert_real_8b_to_float
    : public thrust::unary_function<char,float> {
    inline __host__ __device__
    float operator()(char a) const {
        return (float)a;
    }
};
struct compute_complex_power
    : public thrust::unary_function<float2,float> {
    inline __host__ __device__
    float operator()(float2 a) const {
        return a.x*a.x + a.y*a.y;
    }
};
struct advance_within_region
    : public thrust::unary_function<int,int> {
    int  delta;
    uint region_size;
    advance_within_region(int delta_, uint region_size_)
        : delta(delta_), region_size(region_size_) {}
    inline __host__ __device__
    int operator()(int i) const {
        int region = i / region_size;
        int idx    = i % region_size;
        idx += delta;
        idx = max(0, idx);
        idx = min(region_size-1, idx);
        return idx + region_size*region;
    }
};
struct running_mean_by_region
    : public thrust::unary_function<int, float> {
    uint         radius;
    uint         region_size;
    const float* d_scanned;
    running_mean_by_region(uint radius_,
                           uint region_size_,
                           const float* d_scanned_)
        : radius(radius_),
          region_size(region_size_),
          d_scanned(d_scanned_) {}
    inline __host__ __device__
    float operator()(uint i) const {
        uint region = i / region_size;
        uint offset = region * region_size;
        uint idx    = i % region_size;

        float sum;
        if( idx < radius ) {
            sum = (d_scanned[2*radius + offset] -
                   d_scanned[0 + offset]);
        }
        else if( idx > region_size-1-radius ) {
            sum = (d_scanned[region_size-1 + offset] -
                   d_scanned[region_size-1-2*radius + offset]);
        }
        else {
            sum = (d_scanned[idx + radius + offset] -
                   d_scanned[idx - radius + offset]);
        }
        return sum / (2*radius);
    }
};
struct transpose_index : public thrust::unary_function<size_t,size_t> {
// convert a linear index to a linear index in the transpose 
  size_t m, n;

  __host__ __device__
  transpose_index(size_t _m, size_t _n) : m(_m), n(_n) {}

  __host__ __device__
  size_t operator()(size_t linear_index)
  {
      size_t i = linear_index / n;
      size_t j = linear_index % n;

      return m * j + i;
  }
};
// --------
template<typename T>
struct divide_by : public thrust::unary_function<T,T> {
    T val;
    divide_by(T val_) : val(val_) {}
    inline __host__ __device__
    T operator()(T x) const {
        return x / val;
    }
};
template<typename T>
struct greater_than_val : public thrust::unary_function<T,bool> {
    T val;
    greater_than_val(T val_) : val(val_) {}
    inline __host__ __device__
    bool operator()(T x) const {
        return x > val;
    }
};
template <typename T>
void transpose(size_t m, size_t n, thrust::device_vector<T> *src, thrust::device_vector<T> *dst) {
// transpose an m-by-n array
  thrust::counting_iterator<size_t> indices(0);
  
  thrust::gather
    (thrust::make_transform_iterator(indices, transpose_index(n, m)),
     thrust::make_transform_iterator(indices, transpose_index(n, m)) + dst->size(),
     src->begin(),
     dst->begin());
}

// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};

template <typename Iterator>
class strided_range
{
    public:

    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

    struct stride_functor : public thrust::unary_function<difference_type,difference_type>
    {
        difference_type stride;

        stride_functor(difference_type stride)
            : stride(stride) {}

        __host__ __device__
        difference_type operator()(const difference_type& i) const
        { 
            return stride * i;
        }
    };

    typedef typename thrust::counting_iterator<difference_type>                   CountingIterator;
    typedef typename thrust::transform_iterator<stride_functor, CountingIterator> TransformIterator;
    typedef typename thrust::permutation_iterator<Iterator,TransformIterator>     PermutationIterator;

    // type of the strided_range iterator
    typedef PermutationIterator iterator;

    // construct strided_range for the range [first,last)
    strided_range(Iterator first, Iterator last, difference_type stride)
        : first(first), last(last), stride(stride) {}
   
    iterator begin(void) const
    {
        return PermutationIterator(first, TransformIterator(CountingIterator(0), stride_functor(stride)));
    }

    iterator end(void) const
    {
        return begin() + ((last - first) + (stride - 1)) / stride;
    }
    
    protected:
    Iterator first;
    Iterator last;
    difference_type stride;
};

void do_fft(hipfftHandle *fft_plan, float2* &fft_input_ptr, float2* &fft_output_ptr) {
    Stopwatch timer;
    if(use_timer) timer_start(timer);
    execute_fft_plan_c2c(fft_plan, fft_input_ptr, fft_output_ptr);
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "FFT execution time");
}
void do_r2c_fft(hipfftHandle *fft_plan, float* &fft_input_ptr, float2* &fft_output_ptr) {
    Stopwatch timer;
    if(use_timer) timer_start(timer);
    execute_fft_plan_r2c(fft_plan, fft_input_ptr, fft_output_ptr);
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "FFT execution time");
}

void compute_power_spectrum(device_vectors_t *dv_p) {
//fprintf(stderr, "In compute_power_spectrum 1\n");
    Stopwatch timer;
    if(use_timer) timer_start(timer);
//fprintf(stderr, "In compute_power_spectrum 2 %p %p\n", thrust::raw_pointer_cast(dv_p->fft_data_out_p), thrust::raw_pointer_cast(dv_p->powspec_p));
//fprintf(stderr, "In compute_power_spectrum 2 %p %lu %p %lu\n", dv_p->fft_data_out_p, dv_p->fft_data_out_p->size() * sizeof(float2), dv_p->powspec_p, dv_p->powspec_p->size() * sizeof(float));
	// Here we throw away (the -1) the "padding" element required on the output of the R2C FFT
    thrust::transform(dv_p->fft_data_out_p->begin(), dv_p->fft_data_out_p->end()-1,
                      dv_p->powspec_p->begin(),
                      compute_complex_power());
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Power spectrum time");
//fprintf(stderr, "In compute_power_spectrum 3\n");
}

struct printf_functor {
    __host__ __device__
    void operator()(float x)
    {
      // note that using printf in a __device__ function requires
      // code compiled for a GPU with compute capability 2.0 or
      // higher (nvcc --arch=sm_20)
      printf("%f\n", x);
    }
};

using namespace thrust::placeholders;
void reduce_power_spectra(device_vectors_t *dv_p, int n_subband_pols, int n_chan) {

    // first, sum all of the fine (time) channels for each coarse channel (subband_pol)
    thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0),    // beginning of the input key range 
                            linear_index_to_row_index<int>(n_chan)),                            //  (keyed by row (spectra) index)
                          thrust::make_transform_iterator(thrust::counting_iterator<int>(0),    // end of the input key range
                            linear_index_to_row_index<int>(n_chan)) + (n_subband_pols*n_chan),      
                          dv_p->powspec_p->begin(),                                             // beginning of the input (spectra) value range
                          dv_p->spectra_indices_p->begin(),                                     // beginning of the output (power sums) key range
                          dv_p->spectra_sums_p->begin(),                                        // beginning of the output (power sums) value range
                          thrust::equal_to<int>(),                                              // binary predicate used to determine equality of key
                          thrust::plus<float>());                                               // binary function used to accumulate values
    //thrust::for_each(dv_p->spectra_sums_p->begin(), dv_p->spectra_sums_p->end(), printf_functor());
    // now find the mean of each (TODO why won't the divide_by functor work here?)
    thrust::for_each(dv_p->spectra_sums_p->begin(), dv_p->spectra_sums_p->end(), _1 /= n_chan);
    //thrust::for_each(dv_p->spectra_sums_p->begin(), dv_p->spectra_sums_p->end(), printf_functor());
}

void compute_baseline(device_vectors_t *dv_p, int n_fc, int n_element, float smooth_scale) {
// Compute smoothed power spectrum baseline

    using thrust::make_transform_iterator;
    using thrust::make_counting_iterator;

    if(track_gpu_memory) get_gpu_mem_info("in compute_baseline(), right after making iterators");
    Stopwatch timer;
    if(use_timer) timer_start(timer);
    thrust::exclusive_scan_by_key(make_transform_iterator(make_counting_iterator<int>(0),
                                                          //_1 / n_fc),
                                                          divide_by<int>(n_fc)),
                                  make_transform_iterator(make_counting_iterator<int>(n_element),
                                                          //_1 / n_fc),
                                                          divide_by<int>(n_fc)),
                                  dv_p->powspec_p->begin(),
                                  dv_p->scanned_p->begin());
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Scan time");
    if(track_gpu_memory) get_gpu_mem_info("in compute_baseline(), right after scan");
    
    if(use_timer) timer_start(timer);
    const float* d_scanned_ptr = thrust::raw_pointer_cast(&(*dv_p->scanned_p)[0]);
  //const float* d_scanned_ptr = thrust::raw_pointer_cast(&(*dv.scanned_p   )[0]);
    thrust::transform(make_counting_iterator<uint>(0),
                      make_counting_iterator<uint>(n_element),
                      dv_p->baseline_p->begin(),
                      running_mean_by_region(smooth_scale,
                                             n_fc,
                                             d_scanned_ptr));
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Running mean time");
    if(track_gpu_memory) get_gpu_mem_info("in compute_baseline(), right after running mean by region");
    //thrust::for_each(dv_p->baseline_p->begin(), dv_p->baseline_p->end(), printf_functor());
}

void normalize_power_spectrum(device_vectors_t *dv_p) {

    Stopwatch timer;
    if(use_timer) timer_start(timer);
    thrust::transform(dv_p->powspec_p->begin(), dv_p->powspec_p->end(),
                      dv_p->baseline_p->begin(),
                      dv_p->normalised_p->begin(),
                      //_1 / _2);
                      thrust::divides<float>());
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Normalisation time");
}

size_t find_hits(device_vectors_t *dv_p, int n_element, size_t maxhits, float power_thresh) {
// Extract and retrieve values exceeding the threshold

    using thrust::make_counting_iterator;

    size_t nhits;

    Stopwatch timer;
    if(use_timer) timer_start(timer);
    dv_p->hit_indices_p->resize(n_element); // Note: Upper limit on required storage TODO - is n_element being set right?

    // Find normalised powers (S/N) over threshold.
    // The hit_indices vector will then index the powspec (detected powers) and baseline (mean powers) as well
    // as the normalized power (S/N) vector.
    nhits = thrust::copy_if(make_counting_iterator<int>(0),
                                   make_counting_iterator<int>(n_element),
                                   dv_p->normalised_p->begin(),  // stencil
                                   dv_p->hit_indices_p->begin(), // result
                                   //_1 > power_thresh) - dv_p->hit_indices_p->begin();
                                   greater_than_val<float>(power_thresh))
                                                          - dv_p->hit_indices_p->begin();

    nhits = nhits > maxhits ? maxhits : nhits;       // overrun protection - hits beyond maxgpuhits are thrown away
    dv_p->hit_indices_p->resize(nhits);                 // this will only be resized downwards
                                            
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Hit extraction time");
    
    if(use_timer) timer_start(timer);
    // Retrieve (hit) detected and mean powers into their own vectors for ease of outputting.
    dv_p->hit_powers_p->resize(nhits);
    thrust::gather(dv_p->hit_indices_p->begin(), dv_p->hit_indices_p->end(),
                   dv_p->powspec_p->begin(),
                   dv_p->hit_powers_p->begin());
    dv_p->hit_baselines_p->resize(nhits);
    thrust::gather(dv_p->hit_indices_p->begin(), dv_p->hit_indices_p->end(),
                   dv_p->baseline_p->begin(),
                   dv_p->hit_baselines_p->begin());
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Hit info gather time");

    return nhits;
}    

#ifdef SOURCE_FAST
#if 0
int reduce_coarse_channels(device_vectors_t * dv_p, 
                           s6_output_block_t *s6_output_block,  
                           int n_cc, 
                           int pol, 
                           int n_fc, 
                           int bors) {

    Stopwatch timer;

    if(use_timer) timer_start(timer);

    // allocate working vectors to accomodate all power spectra for this block :
    // all coarse channels (n_cc) x 1 pol
    dv_p->spectra_sums_p      = new thrust::device_vector<float>(n_cc);
    dv_p->spectra_indices_p   = new thrust::device_vector<int>(n_cc);
    if(track_gpu_memory) get_gpu_mem_info("right after vector allocation for coarse channel reduction");

    // do the reduce
    reduce_power_spectra(dv_p, n_cc, n_fc);
    if(track_gpu_memory) get_gpu_mem_info("right after coarse channel reduction");

    // copy the result to the output buffer. Easy copy with just one pol - no strided ranges.
	if(pol == 0) {
    	thrust::copy(dv_p->spectra_sums_p->begin(), dv_p->spectra_sums_p->end(), &(s6_output_block->cc_pwrs_x[bors][0]));
	} else if (pol == 1) {
    	thrust::copy(dv_p->spectra_sums_p->begin(), dv_p->spectra_sums_p->end(), &(s6_output_block->cc_pwrs_y[bors][0]));
	} else {
		fprintf(stderr, "In reduce_coarse_channels() - bad pol index!\n");
	}

    // delete working vectors
    delete(dv_p->spectra_sums_p);
    delete(dv_p->spectra_indices_p);
    if(track_gpu_memory) get_gpu_mem_info("right after vector deletion for coarse channel reduction");

    if(use_timer) sum_of_times += timer_stop(timer, "Reduce coarse channels time");

    return(0);
}
#endif

#else

int reduce_coarse_channels(device_vectors_t * dv_p, 
                           s6_output_block_t *s6_output_block,  
                           int n_cc, 
                           int n_pol, 
                           int n_fc, 
                           int bors) {

    Stopwatch timer;

    if(use_timer) timer_start(timer);

    // allocate working vectors to accomodate all power spectra for this block :
    // all coarse channels (n_cc) x both pols (n_pol)
    dv_p->spectra_sums_p      = new thrust::device_vector<float>(n_cc*n_pol);
    dv_p->spectra_indices_p   = new thrust::device_vector<int>(n_cc*n_pol);
    if(track_gpu_memory) get_gpu_mem_info("right after vector allocation for coarse channel reduction");

    // do the reduce
    reduce_power_spectra(dv_p, n_cc*n_pol, n_fc);
    if(track_gpu_memory) get_gpu_mem_info("right after coarse channel reduction");

    // copy the result to the output buffer, separating the pols. First, create the 
    // strided ranges (2 pols, so a stride of 2) then copy to the output block area
    // for this bors. Note: the "begin() + 1" is to get to the Y pol. 
    typedef thrust::device_vector<float>::iterator Iterator;
    strided_range<Iterator> polX(dv_p->spectra_sums_p->begin(),     dv_p->spectra_sums_p->end(), 2);
    strided_range<Iterator> polY(dv_p->spectra_sums_p->begin() + 1, dv_p->spectra_sums_p->end(), 2);
    thrust::copy(polX.begin(), polX.end(), &(s6_output_block->cc_pwrs_x[bors][0]));
    thrust::copy(polY.begin(), polY.end(), &(s6_output_block->cc_pwrs_y[bors][0]));

    // delete working vectors
    delete(dv_p->spectra_sums_p);
    delete(dv_p->spectra_indices_p);
    if(track_gpu_memory) get_gpu_mem_info("right after vector deletion for coarse channel reduction");

    if(use_timer) sum_of_times += timer_stop(timer, "Reduce coarse channels time");

    return(0);
}

#endif

// AO spectra order goes as pol0chan0 pol0chan1    pol1chan0 pol1chan1    pol0chan2 pol0chan3    pol1chan2 pol1chan3... 
// (S0-C0-P0-Re), (S0-C0-P0-Im), (S0-C1-P0-Re), (S0-C1-P0-Im), (S0-C0-P1-Re), (S0-C0-P1-Im), (S0-C1-P1-Re), (S0-C1-P1-Im)
// foreach spectra
// 	foreach pair of channels
// 		for each pol
// 			8 bits Re, 8 bits Im
inline int ao_pol(long spectrum_index) {
    return((long)floor((double)spectrum_index/2) % 2);
}
inline int ao_coarse_chan(long spectrum_index) {
    return((long)floor((double)spectrum_index/4) * 2 + spectrum_index % 2);
}
// DiBAS (GBT) spectra order goes as pol0chan0 pol1chan0    pol0chan1 pol1chan1    pol0chan2 pol1chan3    pol0chan3 pol1chan3... 
// (S0-C0-P0-Re ), (S0-C0-P0-Im), (S0-C0-P1-Re), (S0-C0-P1-Im), (S0-C1-P0-Re), (S0-C1-P0-Im), (S0-C1-P1-Re), (S0-C1-P1-Im)
// foreach spectra
// 	foreach channel
// 		for each pol
// 			8 bits Re, 8 bits Im
inline int dibas_pol(long spectrum_index) {
    return ((long)(double)spectrum_index % 2);
}
inline int dibas_coarse_chan(long spectrum_index, int sub_spectrum_i) {
    return((long)floor((double)spectrum_index/2) + sub_spectrum_i * N_COARSE_CHAN / N_SUBSPECTRA_PER_SPECTRUM);
}

#ifndef SOURCE_FAST
int spectroscopy(int n_cc,         		// N coarse chans
                 int n_fc,       		// N fine chans (== n_ts in this case)
                 int n_ts,       		// N time samples
                 int n_pol,           	// N pols
                 int bors,              // beam or subspectrum
                 size_t maxhits,
                 size_t maxgpuhits,
                 float power_thresh,
                 float smooth_scale,
                 uint64_t * input_data,
                 size_t n_input_data_bytes,
                 s6_output_block_t *s6_output_block,
				 sem_t * gpu_sem) {

// Note - beam or subspectra. Sometimes we are passed a beam's worth of coarse 
// channels (eg, at AO). At other times we are passed a subspectrum of channels  
// (eg, at GBT). In both cases, each course channel runs the full length of fine
// channels.
 
// Note - GPU memory allocation.  Our total memory needs are larger than the
// capcity of our current GPU (GeForce GTX 780 Ti with 3071MB). So we allocate 
// as needed and delete memory as soon as it is no longer needed.

    Stopwatch timer; 
    Stopwatch total_gpu_timer;
    int n_element = n_cc*n_fc*n_pol;	// number of elements in GPU vectors
    size_t nhits;
    //size_t prior_nhits=0;
    size_t total_nhits=0;
	static device_vectors_t *dv_p = NULL;

    if(track_gpu_memory) {
        char comment[256];
        sprintf(comment, "on entry to non-FAST spectroscopy() : n_element = %d n_input_data_bytes = %lu raw_timeseries_length in char2 = %lu", 
                n_element, n_input_data_bytes, N_COARSE_CHAN / N_SUBSPECTRA_PER_SPECTRUM * N_FINE_CHAN * N_POLS_PER_BEAM);
        get_gpu_mem_info((const char *)comment);
    }

	if(!dv_p) dv_p = init_device_vectors(); 

    char2 * h_raw_timeseries = (char2 *)input_data;

    if(use_total_gpu_timer) total_gpu_timer.start();

    // allocate GPU memory for the timeseries, FFTs and power spectra
    dv_p->fft_data_p         = new thrust::device_vector<float2>(n_element);
    dv_p->fft_data_out_p     = new thrust::device_vector<float2>(n_element);
    dv_p->powspec_p          = new thrust::device_vector<float>(n_element);
    dv_p->raw_timeseries_p   = new thrust::device_vector<char2>(N_COARSE_CHAN / N_SUBSPECTRA_PER_SPECTRUM * N_FINE_CHAN * N_POLS_PER_BEAM);

    // Copy to the device
    if(use_timer) timer.start();

//fprintf(stderr, "HERE 1 %p %p %p %p\n", dv_p->fft_data_p, dv_p->fft_data_out_p, dv_p->powspec_p, dv_p->raw_timeseries_p);

    thrust::copy(h_raw_timeseries, h_raw_timeseries + n_input_data_bytes / sizeof(char2),
                 //d_raw_timeseries.begin());
                 dv_p->raw_timeseries_p->begin());
    if(track_gpu_memory) get_gpu_mem_info("right after time series copy");
    if(use_timer) timer.stop();
    sum_of_times += timer.getTime();
    if(use_timer) cout << "H2D time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();

	sem_wait(gpu_sem);

    if(use_timer) timer.start();
    // Unpack from 8-bit to floats
    thrust::transform(dv_p->raw_timeseries_p->begin(), 
                      dv_p->raw_timeseries_p->end(),
                      dv_p->fft_data_p->begin(),
                      convert_complex_8b_to_float());
    if(use_thread_sync) hipDeviceSynchronize();
    if(track_gpu_memory) get_gpu_mem_info("right after 8bit to float transform");
    if(use_timer) timer.stop();
    sum_of_times += timer.getTime();
    if(use_timer) cout << "Unpack time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
    
    // Input pointer varies with input.
    // Output pointer is constant - we reuse the output area for each input.
    // This is not true anymore - we analyze all inputs in one go. These
    // comments and this way of assigning fft_input_ptr and fft_output_ptr
    // are left as is in case we need to go back to one-input-at-a-time.
    float2* fft_input_ptr  = thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[0]));
    float2* fft_output_ptr = thrust::raw_pointer_cast(&((*dv_p->fft_data_out_p)[0]));

    // FFT. We create and destroy the cufft plan each time around in order to
    // conserve the considerable amount of GPU memory that the plan requires. 
    if(use_timer) timer.start();
    create_fft_plan_1d(fft_plan_p, cufft_config.istride, cufft_config.idist, 
                       cufft_config.ostride, cufft_config.odist, cufft_config.nfft_, 
                       cufft_config.nbatch, cufft_config.fft_type);             // plan FFT
    sum_of_times += timer.getTime();
    if(use_timer) cout << "cufft plan time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
    do_fft                      (fft_plan_p, fft_input_ptr, fft_output_ptr);    // compute FFT
    hipfftDestroy(*fft_plan_p);
    if(track_gpu_memory) get_gpu_mem_info("right after FFT");
    compute_power_spectrum      (dv_p);

    // done with the timeseries and FFTs - delete the associated GPU memory
    if(track_gpu_memory) get_gpu_mem_info("right after compute power spectrum");
    delete(dv_p->raw_timeseries_p);         
    delete(dv_p->fft_data_p);         
    delete(dv_p->fft_data_out_p);     
    if(track_gpu_memory) get_gpu_mem_info("right after post power spectrum deletes");

    // reduce coarse channels to mean power...
    reduce_coarse_channels(dv_p, s6_output_block,  n_cc, n_pol, n_fc, bors);

    // Allocate GPU memory for power normalization
    dv_p->baseline_p         = new thrust::device_vector<float>(n_element);
    if(track_gpu_memory) get_gpu_mem_info("right after baseline vector allocation");
    dv_p->normalised_p       = new thrust::device_vector<float>(n_element);
    if(track_gpu_memory) get_gpu_mem_info("right after normalized vector allocation");
    dv_p->scanned_p          = new thrust::device_vector<float>(n_element);
    if(track_gpu_memory) get_gpu_mem_info("right after scanned vector allocation");
    // Power normalization
    compute_baseline            (dv_p, n_fc, n_element, smooth_scale);        
    if(track_gpu_memory) get_gpu_mem_info("right after baseline computation");
    delete(dv_p->scanned_p);          
    if(track_gpu_memory) get_gpu_mem_info("right after scanned vector deletion");
    normalize_power_spectrum    (dv_p);
    if(track_gpu_memory) get_gpu_mem_info("right after spectrum normalization");
    nhits = find_hits           (dv_p, n_element, maxhits, power_thresh);
    if(track_gpu_memory) get_gpu_mem_info("right after find hits");
    // TODO should probably report if nhits == maxgpuhits, ie overflow
    
    // copy to return vector
    nhits = nhits > maxhits ? maxhits : nhits;
    if(use_timer) timer.start();

    total_nhits += nhits;
    s6_output_block->header.nhits[bors] = nhits;
    // We output both detected and mean powers (not S/N).
    thrust::copy(dv_p->hit_powers_p->begin(),    dv_p->hit_powers_p->end(),    &s6_output_block->power[bors][0]);      
    thrust::copy(dv_p->hit_baselines_p->begin(), dv_p->hit_baselines_p->end(), &s6_output_block->baseline[bors][0]);
    thrust::copy(dv_p->hit_indices_p->begin(),   dv_p->hit_indices_p->end(),   &s6_output_block->hit_indices[bors][0]);
    for(size_t i=0; i<nhits; ++i) {
        long hit_index                        = s6_output_block->hit_indices[bors][i]; 
        long spectrum_index                   = (long)floor((double)hit_index/n_fc);
#ifdef SOURCE_S6
        s6_output_block->pol[bors][i]         = ao_pol(spectrum_index);
        s6_output_block->coarse_chan[bors][i] = ao_coarse_chan(spectrum_index);
#elif SOURCE_DIBAS
        s6_output_block->pol[bors][i]         = dibas_pol(spectrum_index);    
        s6_output_block->coarse_chan[bors][i] = dibas_coarse_chan(spectrum_index, bors);
#endif
        s6_output_block->fine_chan[bors][i]   = hit_index % n_fc;
        //fprintf(stderr, "hit_index %ld spectrum_index %ld pol %d cchan %d fchan %d power %f\n", 
        //        hit_index, spectrum_index, s6_output_block->pol[bors][i], s6_output_block->coarse_chan[bors][i], 
        //        s6_output_block->fine_chan[bors][i], s6_output_block->power[bors][i]);
    }
        
    // delete remaining GPU memory
    delete(dv_p->powspec_p);          
    delete(dv_p->baseline_p);         
    delete(dv_p->normalised_p);       
       
    if(use_timer) timer.stop();
    sum_of_times += timer.getTime();
    if(use_timer) cout << "Copy to return vector time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();

    if(use_total_gpu_timer) total_gpu_timer.stop();
    if(use_total_gpu_timer) cout << "Total GPU time:\t" << total_gpu_timer.getTime() << endl;
    if(use_total_gpu_timer) total_gpu_timer.reset();
    
	sem_post(gpu_sem);

    return total_nhits;
}
#endif

#ifdef SOURCE_FAST    
#ifdef REALLOC_CUB
int spectroscopy(int n_cc, 				// N coarse chans
                 int n_fc,    			// N fine chans
                 int n_ts,    			// N time samples
                 int n_pol,           	// N pols
                 int bors,              // beam or subspectrum
                 size_t maxhits,
                 size_t maxgpuhits,
                 float power_thresh,
                 float smooth_scale,
                 uint64_t * input_data,
                 size_t n_input_data_bytes,
                 s6_output_block_t *s6_output_block,
				 sem_t * gpu_sem) {

// Note - beam or subspectra. Sometimes we are passed a beam's worth of coarse 
// channels (eg, at AO). At other times we are passed a subspectrum of channels  
// (eg, at GBT). In both cases, each course channel runs the full length of fine
// channels.
 
// Note - GPU memory allocation.  Our total memory needs are larger than the
// capcity of our current GPU (GeForce GTX 780 Ti with 3071MB). So we allocate 
// as needed and delete memory as soon as it is no longer needed.

    Stopwatch timer; 
    Stopwatch total_gpu_timer;
    Stopwatch mem_timer;
    Stopwatch sem_timer;
    int n_element = n_cc*n_fc;       // number of elements in GPU structures
    size_t nhits;
    size_t total_nhits=0;
    hipfftHandle fft_plan;
    hipfftHandle *fft_plan_p = &fft_plan;
    int pol = n_pol;                // for ease of code reading
	static device_vectors_t *dv_p = NULL;

    sum_of_times=0;
    sum_of_mem_times=0;    
    float sem_time=0;    

	fprintf(stderr, "Reallocating GPU memory via CUB caching allocator\n");

    if(track_gpu_memory) {
        char comment[256];
        sprintf(comment, "on entry to FAST spectroscopy() : n_pol = %d n_element = %d raw_timeseries_length in bytes = %lu (%3.2lf gigasamples) input data located at %p", 
                n_pol, n_element, n_input_data_bytes, (double)n_input_data_bytes/1024/1024/1024, input_data);
        get_gpu_mem_info((const char *)comment);
    }

	if(!dv_p) dv_p = init_device_vectors(); 

    char * h_raw_timeseries = (char *)input_data;

//#define DUMP_RAW_SAMPLES
#ifdef DUMP_RAW_SAMPLES
    static int cnt = 0;
    if(cnt++ == 10) {                                                       // wait for 10 buffers to make sure we are settled
        int num_samples_to_dump = 8*1024;
        for(int i=0; i < num_samples_to_dump; i++) printf("%d\n", h_raw_timeseries[i]);   
    }
#endif

    if(use_total_gpu_timer) total_gpu_timer.start();

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->raw_timeseries_p   = new thrust::device_vector<char>(n_input_data_bytes);  
    dv_p->raw_timeseries_p   = new cub_device_vector<char>(n_input_data_bytes);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new raw_timeseries time");

    // Copy to the device
//print_current_time("right before time series copy");
    if(use_timer) timer_start(timer);
    thrust::copy(h_raw_timeseries, h_raw_timeseries + n_input_data_bytes / sizeof(char),
                 dv_p->raw_timeseries_p->begin());
    if(use_timer) sum_of_times += timer_stop(timer, "H2D time");
    if(track_gpu_memory) get_gpu_mem_info("right after time series copy");

//print_current_time("right before sem wait");
    if(use_sem_timer) timer_start(sem_timer);
	sem_wait(gpu_sem);
    if(use_sem_timer) sem_time = timer_stop(sem_timer, "sem wait time");
//print_current_time("right after sem wait");

    // allocate (and delete - see below) 
    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->hit_indices_p      = new thrust::device_vector<int>();                        // 0 initial size
    dv_p->hit_indices_p      = new cub_device_vector<int>();                        // 0 initial size
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_indices_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->hit_powers_p       = new thrust::device_vector<float>;                        // "
    dv_p->hit_powers_p       = new cub_device_vector<float>;                        // "
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_powers_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->hit_baselines_p    = new thrust::device_vector<float>;                        // "
    dv_p->hit_baselines_p    = new cub_device_vector<float>;                        // "
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_baselines_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->fft_data_p         = new thrust::device_vector<float>(2*N_FINE_CHAN);    	// if doing the FFT in place (not tested)
    //dv_p->fft_data_p         = new thrust::device_vector<float>(n_ts);         			// FFT input
    dv_p->fft_data_p         = new cub_device_vector<float>(n_ts);         			// FFT input
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new fft_data_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after FFT input vector allocation");

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->fft_data_out_p     = (float2*)dv_p->fft_data_p;                             // if doing the FFT in place (not tested)
    //dv_p->fft_data_out_p     = new thrust::device_vector<float2>(n_element);            // FFT output
    dv_p->fft_data_out_p     = new cub_device_vector<float2>(n_element+1);            // FFT output
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new fft_data_out_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after FFT output vector allocation");


    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->powspec_p = new thrust::device_vector<float>(n_element);             // power spectrum
    dv_p->powspec_p = new cub_device_vector<float>(n_element);             // power spectrum
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new powspec_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after powerspec vector allocation");

    // Unpack from 8-bit to floats
    if(use_timer) timer_start(timer);
    thrust::transform(dv_p->raw_timeseries_p->begin(), 
                  dv_p->raw_timeseries_p->end(),
                  dv_p->fft_data_p->begin(),
                  convert_real_8b_to_float());
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Unpack time");
    if(track_gpu_memory) get_gpu_mem_info("right after 8bit to float transform");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->raw_timeseries_p);   
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete raw_timeseries_p time");
    // end fluffing to FFT input
    
    // Input pointer varies with input.
    // Output pointer is constant - we reuse the output area for each input.
    // This is not true anymore - we analyze all inputs in one go. These
    // comments and this way of assigning fft_input_ptr and fft_output_ptr
    // are left as is in case we need to go back to one-input-at-a-time.
    float*  fft_input_ptr  = thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[0]));
    float2* fft_output_ptr = thrust::raw_pointer_cast(&((*dv_p->fft_data_out_p)[0]));
    //float2* fft_output_ptr = (float2*)thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[0])); // if doing the FFT in place (not tested)

    // FFT. We create and destroy the cufft plan each time around in order to
    // conserve the considerable amount of GPU memory that the plan requires. 
    if(use_timer) timer_start(timer);
    create_fft_plan_1d(fft_plan_p, cufft_config.istride, cufft_config.idist, 
                       cufft_config.ostride, cufft_config.odist, cufft_config.nfft_, 
                       cufft_config.nbatch, cufft_config.fft_type);                 // plan FFT
    if(use_timer) sum_of_times += timer_stop(timer, "cufft plan time");
    do_r2c_fft                      (fft_plan_p, fft_input_ptr, fft_output_ptr);    // compute FFT
    hipfftDestroy(*fft_plan_p);
    if(track_gpu_memory) get_gpu_mem_info("right after FFT");

	//dv_p->fft_data_out_p->erase(dv_p->fft_data_out_p->end());

    compute_power_spectrum      (dv_p);                                         // compute power spectrum

    // done with the timeseries and FFTs - delete the associated GPU memory
    if(track_gpu_memory) get_gpu_mem_info("right after compute power spectrum");
    //delete(dv_p->raw_timeseries_p);   // two pols        
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->fft_data_p);         
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete fft_data_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->fft_data_out_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete fft_data_out_p time");

    if(use_mem_timer) timer_start(mem_timer);
    get_singleton_device_allocator()->free_all_cached();    // free all cub cached allocations
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem free_all_cached 1 time");

    if(track_gpu_memory) get_gpu_mem_info("right after post power spectrum deletes");

    // reduce coarse channels to mean power... we can skip this for FAST
    //reduce_coarse_channels(dv_p, s6_output_block,  n_cc, pol, n_fc, bors);

    // Allocate GPU memory for power normalization
    //dv_p->baseline_p         = new thrust::device_vector<float>(n_element);

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->baseline_p         = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new baseline_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after baseline vector allocation");
    //dv_p->normalised_p       = new thrust::device_vector<float>(n_element);

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->normalised_p       = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new normalised_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after normalized vector allocation");
    //dv_p->scanned_p          = new thrust::device_vector<float>(n_element);

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->scanned_p          = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new scanned_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after scanned vector allocation");

    // Power normalization
    compute_baseline            (dv_p, n_fc, n_element, smooth_scale);     
    if(track_gpu_memory) get_gpu_mem_info("right after baseline computation");
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->scanned_p);          
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete scanned_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after scanned vector deletion");
    normalize_power_spectrum    (dv_p);

    // Hit finding
    if(track_gpu_memory) get_gpu_mem_info("right after spectrum normalization");
    nhits = find_hits           (dv_p, n_element, maxhits, power_thresh);
    if(track_gpu_memory) get_gpu_mem_info("right after find hits");
    // TODO should probably report if nhits == maxgpuhits, ie overflow
    
    // copy to return vector
    nhits = nhits > maxhits ? maxhits : nhits;
    if(use_timer) timer_start(timer);
    total_nhits += nhits;
    s6_output_block->header.nhits[bors] = nhits;
    // We output both detected and mean powers (not S/N).
    thrust::copy(dv_p->hit_powers_p->begin(),    dv_p->hit_powers_p->end(),    &s6_output_block->power[bors][0]);      
    thrust::copy(dv_p->hit_baselines_p->begin(), dv_p->hit_baselines_p->end(), &s6_output_block->baseline[bors][0]);
    thrust::copy(dv_p->hit_indices_p->begin(),   dv_p->hit_indices_p->end(),   &s6_output_block->hit_indices[bors][0]);
    for(size_t i=0; i<nhits; ++i) {
        long hit_index                        = s6_output_block->hit_indices[bors][i]; 
        long spectrum_index                   = (long)floor((double)hit_index/n_fc);
#ifdef SOURCE_S6
        s6_output_block->pol[bors][i]         = ao_pol(spectrum_index);
        s6_output_block->coarse_chan[bors][i] = ao_coarse_chan(spectrum_index);
#elif SOURCE_DIBAS
        s6_output_block->pol[bors][i]         = dibas_pol(spectrum_index);    
        s6_output_block->coarse_chan[bors][i] = dibas_coarse_chan(spectrum_index, bors);
#elif SOURCE_FAST
        s6_output_block->pol[bors][i]         = pol;   
        s6_output_block->coarse_chan[bors][i] = 0;  // 1 coarse channel for FAST, thus cc number is always 0
#endif
        s6_output_block->fine_chan[bors][i]   = hit_index % n_fc;
//#define PRINT_HIT_INFO
#ifdef PRINT_HIT_INFO
        fprintf(stderr, "bors %d i %d hit_index %ld spectrum_index %ld pol %d cchan %d fchan %d power %f\n", 
                bors, i, hit_index, spectrum_index, s6_output_block->pol[bors][i], s6_output_block->coarse_chan[bors][i], 
                s6_output_block->fine_chan[bors][i], s6_output_block->power[bors][i]);
#endif
    } // end for i<nhits 
    if(use_timer) sum_of_times += timer_stop(timer, "Copy to return vector time");
        
    // delete remaining GPU memory
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    delete dv_p->powspec_p;          
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete powspec_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete dv_p->baseline_p;         
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete baseline_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->normalised_p);       
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete nomalised_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->hit_baselines_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_baselines_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->hit_indices_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_indices_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->hit_powers_p); 
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_powers_p time");

    //delete(dv_p->raw_timeseries_p);   

//print_current_time("right after sem post");

    if(use_mem_timer) timer_start(mem_timer);
    get_singleton_device_allocator()->free_all_cached();    // free all cub allocations
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem free_all_cached 2 time");

	sem_post(gpu_sem);

    if(use_total_gpu_timer) total_gpu_timer.stop();
    if(use_total_gpu_timer) cout << "Sum of GPU times:         \t" << sum_of_times << endl;
    if(use_mem_timer)       cout << "Sum of mem times:         \t" << sum_of_mem_times << endl;    
    if(use_sem_timer)       cout << "Sem time:                 \t" << sem_time << endl;    
    if(use_total_gpu_timer) cout << "Uncounted time:           \t" << total_gpu_timer.getTime() - (sum_of_times + sum_of_mem_times + sem_time) << endl;
    if(use_total_gpu_timer) cout << "Total spectroscopy() time:\t" << total_gpu_timer.getTime() << endl;
    if(use_total_gpu_timer) total_gpu_timer.reset();

    cout<<"------------------------------------------------------------------------------------------"<<endl;
    if(track_gpu_memory) get_gpu_mem_info("right before return to gpu thread");
    return total_nhits;
}

#endif
#ifdef REALLOC_STD

int spectroscopy(int n_cc, 				// N coarse chans
                 int n_fc,    			// N fine chans
                 int n_ts,    			// N time samples
                 int n_pol,           	// N pols
                 int bors,              // beam or subspectrum
                 size_t maxhits,
                 size_t maxgpuhits,
                 float power_thresh,
                 float smooth_scale,
                 uint64_t * input_data,
                 size_t n_input_data_bytes,
                 s6_output_block_t *s6_output_block,
				 sem_t * gpu_sem) {

// Note - beam or subspectra. Sometimes we are passed a beam's worth of coarse 
// channels (eg, at AO). At other times we are passed a subspectrum of channels  
// (eg, at GBT). In both cases, each course channel runs the full length of fine
// channels.
 
// Note - GPU memory allocation.  Our total memory needs are larger than the
// capcity of our current GPU (GeForce GTX 780 Ti with 3071MB). So we allocate 
// as needed and delete memory as soon as it is no longer needed.

    Stopwatch timer; 
    Stopwatch total_gpu_timer;
    Stopwatch mem_timer;
    Stopwatch sem_timer;
    int n_element = n_cc*n_fc;       // number of elements in GPU structures
    size_t nhits;
    size_t total_nhits=0;
    hipfftHandle fft_plan;
    hipfftHandle *fft_plan_p = &fft_plan;
    int pol = n_pol;                // for ease of code reading
	static device_vectors_t *dv_p = NULL;

    sum_of_times=0;
    sum_of_mem_times=0;    
    float sem_time=0;    

	fprintf(stderr, "Reallocating GPU memory via standard new/delete\n");

    if(track_gpu_memory) {
        char comment[256];
        sprintf(comment, "on entry to FAST spectroscopy() : n_pol = %d n_element = %d raw_timeseries_length in bytes = %lu (%3.2lf gigasamples) input data located at %p", 
                n_pol, n_element, n_input_data_bytes, (double)n_input_data_bytes/1024/1024/1024, input_data);
        get_gpu_mem_info((const char *)comment);
    }

	if(!dv_p) dv_p = init_device_vectors(); 

    char * h_raw_timeseries = (char *)input_data;

//#define DUMP_RAW_SAMPLES
#ifdef DUMP_RAW_SAMPLES
    static int cnt = 0;
    if(cnt++ == 10) {                                                       // wait for 10 buffers to make sure we are settled
        int num_samples_to_dump = 8*1024;
        for(int i=0; i < num_samples_to_dump; i++) printf("%d\n", h_raw_timeseries[i]);   
    }
#endif

    if(use_total_gpu_timer) total_gpu_timer.start();

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->raw_timeseries_p   = new thrust::device_vector<char>(n_input_data_bytes);  
    //dv_p->raw_timeseries_p   = new cub_device_vector<char>(n_input_data_bytes);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new raw_timeseries time");

    // Copy to the device
//print_current_time("right before time series copy");
    if(use_timer) timer_start(timer);
    thrust::copy(h_raw_timeseries, h_raw_timeseries + n_input_data_bytes / sizeof(char),
                 dv_p->raw_timeseries_p->begin());
    if(use_timer) sum_of_times += timer_stop(timer, "H2D time");
    if(track_gpu_memory) get_gpu_mem_info("right after time series copy");

//print_current_time("right before sem wait");
    if(use_sem_timer) timer_start(sem_timer);
	sem_wait(gpu_sem);
    if(use_sem_timer) sem_time = timer_stop(sem_timer, "sem wait time");
//print_current_time("right after sem wait");

    // allocate (and delete - see below) 
    if(use_mem_timer) timer_start(mem_timer);
    dv_p->hit_indices_p      = new thrust::device_vector<int>();                        // 0 initial size
    //dv_p->hit_indices_p      = new cub_device_vector<int>();                        // 0 initial size
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_indices_p time");

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->hit_powers_p       = new thrust::device_vector<float>;                        // "
    //dv_p->hit_powers_p       = new cub_device_vector<float>;                        // "
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_powers_p time");

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->hit_baselines_p    = new thrust::device_vector<float>;                        // "
    //dv_p->hit_baselines_p    = new cub_device_vector<float>;                        // "
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_baselines_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->fft_data_p         = new thrust::device_vector<float>(2*N_FINE_CHAN);    	// if doing the FFT in place (not tested)
    dv_p->fft_data_p         = new thrust::device_vector<float>(n_ts);         			// FFT input
    //dv_p->fft_data_p         = new cub_device_vector<float>(n_ts);         			// FFT input
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new fft_data_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after FFT input vector allocation");

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->fft_data_out_p     = (float2*)dv_p->fft_data_p;                             // if doing the FFT in place (not tested)
    dv_p->fft_data_out_p     = new thrust::device_vector<float2>(n_element);            // FFT output
    //dv_p->fft_data_out_p     = new cub_device_vector<float2>(n_element+1);            // FFT output
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new fft_data_out_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after FFT output vector allocation");


    if(use_mem_timer) timer_start(mem_timer);
    dv_p->powspec_p = new thrust::device_vector<float>(n_element);             // power spectrum
    //dv_p->powspec_p = new cub_device_vector<float>(n_element);             // power spectrum
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new powspec_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after powerspec vector allocation");

    // Unpack from 8-bit to floats
    if(use_timer) timer_start(timer);
    thrust::transform(dv_p->raw_timeseries_p->begin(), 
                  dv_p->raw_timeseries_p->end(),
                  dv_p->fft_data_p->begin(),
                  convert_real_8b_to_float());
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Unpack time");
    if(track_gpu_memory) get_gpu_mem_info("right after 8bit to float transform");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->raw_timeseries_p);   
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete raw_timeseries_p time");
    // end fluffing to FFT input
    
    // Input pointer varies with input.
    // Output pointer is constant - we reuse the output area for each input.
    // This is not true anymore - we analyze all inputs in one go. These
    // comments and this way of assigning fft_input_ptr and fft_output_ptr
    // are left as is in case we need to go back to one-input-at-a-time.
    float*  fft_input_ptr  = thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[0]));
    float2* fft_output_ptr = thrust::raw_pointer_cast(&((*dv_p->fft_data_out_p)[0]));
    //float2* fft_output_ptr = (float2*)thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[0])); // if doing the FFT in place (not tested)

    // FFT. We create and destroy the cufft plan each time around in order to
    // conserve the considerable amount of GPU memory that the plan requires. 
    if(use_timer) timer_start(timer);
    create_fft_plan_1d(fft_plan_p, cufft_config.istride, cufft_config.idist, 
                       cufft_config.ostride, cufft_config.odist, cufft_config.nfft_, 
                       cufft_config.nbatch, cufft_config.fft_type);                 // plan FFT
    if(use_timer) sum_of_times += timer_stop(timer, "cufft plan time");
    do_r2c_fft                      (fft_plan_p, fft_input_ptr, fft_output_ptr);    // compute FFT
    hipfftDestroy(*fft_plan_p);
    if(track_gpu_memory) get_gpu_mem_info("right after FFT");

	//dv_p->fft_data_out_p->erase(dv_p->fft_data_out_p->end());

    compute_power_spectrum      (dv_p);                                         // compute power spectrum

    // done with the timeseries and FFTs - delete the associated GPU memory
    if(track_gpu_memory) get_gpu_mem_info("right after compute power spectrum");
    //delete(dv_p->raw_timeseries_p);   // two pols        
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->fft_data_p);         
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete fft_data_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->fft_data_out_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete fft_data_out_p time");

    //if(use_mem_timer) timer_start(mem_timer);
    //get_singleton_device_allocator()->free_all_cached();    // free all cub cached allocations
    //if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem free_all_cached 1 time");

    if(track_gpu_memory) get_gpu_mem_info("right after post power spectrum deletes");

    // reduce coarse channels to mean power... we can skip this for FAST
    //reduce_coarse_channels(dv_p, s6_output_block,  n_cc, pol, n_fc, bors);

    // Allocate GPU memory for power normalization

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->baseline_p         = new thrust::device_vector<float>(n_element);
    //dv_p->baseline_p         = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new baseline_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after baseline vector allocation");

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->normalised_p       = new thrust::device_vector<float>(n_element);
    //dv_p->normalised_p       = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new normalised_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after normalized vector allocation");

    if(use_mem_timer) timer_start(mem_timer);
    dv_p->scanned_p          = new thrust::device_vector<float>(n_element);
    //dv_p->scanned_p          = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new scanned_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after scanned vector allocation");

    // Power normalization
    compute_baseline            (dv_p, n_fc, n_element, smooth_scale);     
    if(track_gpu_memory) get_gpu_mem_info("right after baseline computation");
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->scanned_p);          
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete scanned_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after scanned vector deletion");
    normalize_power_spectrum    (dv_p);

    // Hit finding
    if(track_gpu_memory) get_gpu_mem_info("right after spectrum normalization");
    nhits = find_hits           (dv_p, n_element, maxhits, power_thresh);
    if(track_gpu_memory) get_gpu_mem_info("right after find hits");
    // TODO should probably report if nhits == maxgpuhits, ie overflow
    
    // copy to return vector
    nhits = nhits > maxhits ? maxhits : nhits;
    if(use_timer) timer_start(timer);
    total_nhits += nhits;
    s6_output_block->header.nhits[bors] = nhits;
    // We output both detected and mean powers (not S/N).
    thrust::copy(dv_p->hit_powers_p->begin(),    dv_p->hit_powers_p->end(),    &s6_output_block->power[bors][0]);      
    thrust::copy(dv_p->hit_baselines_p->begin(), dv_p->hit_baselines_p->end(), &s6_output_block->baseline[bors][0]);
    thrust::copy(dv_p->hit_indices_p->begin(),   dv_p->hit_indices_p->end(),   &s6_output_block->hit_indices[bors][0]);
    for(size_t i=0; i<nhits; ++i) {
        long hit_index                        = s6_output_block->hit_indices[bors][i]; 
        long spectrum_index                   = (long)floor((double)hit_index/n_fc);
#ifdef SOURCE_S6
        s6_output_block->pol[bors][i]         = ao_pol(spectrum_index);
        s6_output_block->coarse_chan[bors][i] = ao_coarse_chan(spectrum_index);
#elif SOURCE_DIBAS
        s6_output_block->pol[bors][i]         = dibas_pol(spectrum_index);    
        s6_output_block->coarse_chan[bors][i] = dibas_coarse_chan(spectrum_index, bors);
#elif SOURCE_FAST
        s6_output_block->pol[bors][i]         = pol;   
        s6_output_block->coarse_chan[bors][i] = 0;  // 1 coarse channel for FAST, thus cc number is always 0
#endif
        s6_output_block->fine_chan[bors][i]   = hit_index % n_fc;
//#define PRINT_HIT_INFO
#ifdef PRINT_HIT_INFO
        fprintf(stderr, "bors %d i %d hit_index %ld spectrum_index %ld pol %d cchan %d fchan %d power %f\n", 
                bors, i, hit_index, spectrum_index, s6_output_block->pol[bors][i], s6_output_block->coarse_chan[bors][i], 
                s6_output_block->fine_chan[bors][i], s6_output_block->power[bors][i]);
#endif
    } // end for i<nhits 
    if(use_timer) sum_of_times += timer_stop(timer, "Copy to return vector time");
        
    // delete remaining GPU memory
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    delete dv_p->powspec_p;          
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete powspec_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete dv_p->baseline_p;         
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete baseline_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->normalised_p);       
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete nomalised_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->hit_baselines_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_baselines_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->hit_indices_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_indices_p time");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->hit_powers_p); 
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_powers_p time");

    //delete(dv_p->raw_timeseries_p);   

//print_current_time("right after sem post");

    //if(use_mem_timer) timer_start(mem_timer);
    //get_singleton_device_allocator()->free_all_cached();    // free all cub allocations
   	//if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem free_all_cached 2 time");

	sem_post(gpu_sem);

    if(use_total_gpu_timer) total_gpu_timer.stop();
    if(use_total_gpu_timer) cout << "Sum of GPU times:         \t" << sum_of_times << endl;
    if(use_mem_timer)       cout << "Sum of mem times:         \t" << sum_of_mem_times << endl;    
    if(use_sem_timer)       cout << "Sem time:                 \t" << sem_time << endl;    
    if(use_total_gpu_timer) cout << "Uncounted time:           \t" << total_gpu_timer.getTime() - (sum_of_times + sum_of_mem_times + sem_time) << endl;
    if(use_total_gpu_timer) cout << "Total spectroscopy() time:\t" << total_gpu_timer.getTime() << endl;
    if(use_total_gpu_timer) total_gpu_timer.reset();

    cout<<"------------------------------------------------------------------------------------------"<<endl;
    if(track_gpu_memory) get_gpu_mem_info("right before return to gpu thread");
    return total_nhits;
}

#endif
#ifdef REALLOC_NONE

int spectroscopy(int n_cc, 				// N coarse chans
                 int n_fc,    			// N fine chans
                 int n_ts,    			// N time samples
                 int n_pol,           	// N pols
                 int bors,              // beam or subspectrum
                 size_t maxhits,
                 size_t maxgpuhits,
                 float power_thresh,
                 float smooth_scale,
                 uint64_t * input_data,
                 size_t n_input_data_bytes,
                 s6_output_block_t *s6_output_block,
				 sem_t * gpu_sem) {

// Note - beam or subspectra. Sometimes we are passed a beam's worth of coarse 
// channels (eg, at AO). At other times we are passed a subspectrum of channels  
// (eg, at GBT). In both cases, each course channel runs the full length of fine
// channels.
 
// Note - this version does minimal GPU memory re-allocation.  Our total memory 
// needs are larger than the capcity of our current GPU (GeForce GTX 780 Ti with 
// 3071MB). So we allocate as needed and delete memory as soon as it is no longer needed.

    Stopwatch timer; 
    Stopwatch total_gpu_timer;
    Stopwatch mem_timer;
    Stopwatch sem_timer;
    int n_element = n_cc*n_fc;       // number of elements in GPU structures
    size_t nhits;
    size_t total_nhits=0;
    hipfftHandle fft_plan;
    hipfftHandle *fft_plan_p = &fft_plan;
    //static hipfftHandle fft_plan;
    //static hipfftHandle *fft_plan_p = &fft_plan;
    int pol = n_pol;                // for ease of code reading
	static device_vectors_t *dv_p = NULL;

    sum_of_times=0;
    sum_of_mem_times=0;    
    float sem_time=0;    

	//fprintf(stderr, "Not reallocating GPU memory\n");

    if(track_gpu_memory) {
        char comment[256];
        sprintf(comment, "on entry to FAST spectroscopy() : n_pol = %d n_element = %d raw_timeseries_length in bytes = %lu (%3.2lf gigasamples) input data located at %p", 
                n_pol, n_element, n_input_data_bytes, (double)n_input_data_bytes/1024/1024/1024, input_data);
        get_gpu_mem_info((const char *)comment);
    }

	if(!dv_p) dv_p = init_device_vectors(); 

    char * h_raw_timeseries = (char *)input_data;

//#define DUMP_RAW_SAMPLES
#ifdef DUMP_RAW_SAMPLES
    static int cnt = 0;
    if(cnt++ == 10) {                                                       // wait for 10 buffers to make sure we are settled
        int num_samples_to_dump = 8*1024;
        for(int i=0; i < num_samples_to_dump; i++) printf("%d\n", h_raw_timeseries[i]);   
    }
#endif

    if(use_total_gpu_timer) total_gpu_timer.start();

    if(use_mem_timer) timer_start(mem_timer);
    if(!dv_p->raw_timeseries_p) dv_p->raw_timeseries_p   = new thrust::device_vector<char>(n_input_data_bytes);  
    //dv_p->raw_timeseries_p   = new cub_device_vector<char>(n_input_data_bytes);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new raw_timeseries time");

    // Copy to the device
//print_current_time("right before time series copy");
    if(use_timer) timer_start(timer);
    thrust::copy(h_raw_timeseries, h_raw_timeseries + n_input_data_bytes / sizeof(char),
                 dv_p->raw_timeseries_p->begin());
    if(use_timer) sum_of_times += timer_stop(timer, "H2D time");
    if(track_gpu_memory) get_gpu_mem_info("right after time series copy");

//print_current_time("right before sem wait");
    if(use_sem_timer) timer_start(sem_timer);
	sem_wait(gpu_sem);
    if(use_sem_timer) sem_time = timer_stop(sem_timer, "sem wait time");
//print_current_time("right after sem wait");

    // allocate (and delete - see below) 
    if(use_mem_timer) timer_start(mem_timer);
    if(!dv_p->hit_indices_p) dv_p->hit_indices_p      = new thrust::device_vector<int>();                        // 0 initial size
    //dv_p->hit_indices_p      = new cub_device_vector<int>();                        // 0 initial size
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_indices_p time");

    if(use_mem_timer) timer_start(mem_timer);
    if(!dv_p->hit_powers_p) dv_p->hit_powers_p       = new thrust::device_vector<float>;                        // "
    //dv_p->hit_powers_p       = new cub_device_vector<float>;                        // "
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_powers_p time");

    if(use_mem_timer) timer_start(mem_timer);
    if(!dv_p->hit_baselines_p) dv_p->hit_baselines_p    = new thrust::device_vector<float>;                        // "
    //dv_p->hit_baselines_p    = new cub_device_vector<float>;                        // "
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new hit_baselines_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->fft_data_p         = new thrust::device_vector<float>(2*N_FINE_CHAN);    	// if doing the FFT in place (not tested)
    if(!dv_p->fft_data_p) dv_p->fft_data_p         = new thrust::device_vector<float>(n_ts);         			// FFT input
    //dv_p->fft_data_p         = new cub_device_vector<float>(n_ts);         			// FFT input
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new fft_data_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after FFT input vector allocation");

    if(use_mem_timer) timer_start(mem_timer);
    //dv_p->fft_data_out_p     = (float2*)dv_p->fft_data_p;                             // if doing the FFT in place (not tested)
    if(!dv_p->fft_data_out_p) dv_p->fft_data_out_p     = new thrust::device_vector<float2>(n_element);            // FFT output
    //dv_p->fft_data_out_p     = new cub_device_vector<float2>(n_element+1);            // FFT output
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new fft_data_out_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after FFT output vector allocation");


    if(use_mem_timer) timer_start(mem_timer);
    if(!dv_p->powspec_p) dv_p->powspec_p = new thrust::device_vector<float>(n_element);             // power spectrum
    //dv_p->powspec_p = new cub_device_vector<float>(n_element);             // power spectrum
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new powspec_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after powerspec vector allocation");

    // Unpack from 8-bit to floats
    if(use_timer) timer_start(timer);
    thrust::transform(dv_p->raw_timeseries_p->begin(), 
                  dv_p->raw_timeseries_p->end(),
                  dv_p->fft_data_p->begin(),
                  convert_real_8b_to_float());
    if(use_thread_sync) hipDeviceSynchronize();
    if(use_timer) sum_of_times += timer_stop(timer, "Unpack time");
    if(track_gpu_memory) get_gpu_mem_info("right after 8bit to float transform");

    if(use_mem_timer) timer_start(mem_timer);
    delete(dv_p->raw_timeseries_p); dv_p->raw_timeseries_p = 0;   
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete raw_timeseries_p time");
    // end fluffing to FFT input
    
    // Input pointer varies with input.
    // Output pointer is constant - we reuse the output area for each input.
    // This is not true anymore - we analyze all inputs in one go. These
    // comments and this way of assigning fft_input_ptr and fft_output_ptr
    // are left as is in case we need to go back to one-input-at-a-time.
    float*  fft_input_ptr  = thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[0]));
    float2* fft_output_ptr = thrust::raw_pointer_cast(&((*dv_p->fft_data_out_p)[0]));
    //float2* fft_output_ptr = (float2*)thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[0])); // if doing the FFT in place (not tested)

    // FFT. We create and destroy the cufft plan each time around in order to
    // conserve the considerable amount of GPU memory that the plan requires. 
   	if(use_timer) timer_start(timer);
   		create_fft_plan_1d(fft_plan_p, cufft_config.istride, cufft_config.idist, 
                       cufft_config.ostride, cufft_config.odist, cufft_config.nfft_, 
                       cufft_config.nbatch, cufft_config.fft_type);                 // plan FFT
   	if(use_timer) sum_of_times += timer_stop(timer, "cufft plan time");
    do_r2c_fft                      (fft_plan_p, fft_input_ptr, fft_output_ptr);    // compute FFT
    hipfftDestroy(*fft_plan_p);
    if(track_gpu_memory) get_gpu_mem_info("right after FFT");

	//dv_p->fft_data_out_p->erase(dv_p->fft_data_out_p->end());

    compute_power_spectrum      (dv_p);                                         // compute power spectrum

    // done with the timeseries and FFTs - delete the associated GPU memory
    if(track_gpu_memory) get_gpu_mem_info("right after compute power spectrum");
    //delete(dv_p->raw_timeseries_p);   // two pols        
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    //..delete(dv_p->fft_data_p);         
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete fft_data_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //..delete(dv_p->fft_data_out_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete fft_data_out_p time");

    //if(use_mem_timer) timer_start(mem_timer);
    //get_singleton_device_allocator()->free_all_cached();    // free all cub cached allocations
    //if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem free_all_cached 1 time");

    if(track_gpu_memory) get_gpu_mem_info("right after post power spectrum deletes");

    // reduce coarse channels to mean power... we can skip this for FAST
    //reduce_coarse_channels(dv_p, s6_output_block,  n_cc, pol, n_fc, bors);

    // Allocate GPU memory for power normalization

    if(use_mem_timer) timer_start(mem_timer);
    if(!dv_p->baseline_p) dv_p->baseline_p         = new thrust::device_vector<float>(n_element);
    //dv_p->baseline_p         = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new baseline_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after baseline vector allocation");

    if(use_mem_timer) timer_start(mem_timer);
    if(!dv_p->normalised_p) dv_p->normalised_p       = new thrust::device_vector<float>(n_element);
    //dv_p->normalised_p       = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new normalised_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after normalized vector allocation");

    if(use_mem_timer) timer_start(mem_timer);
    if(!dv_p->scanned_p) dv_p->scanned_p          = new thrust::device_vector<float>(n_element);
    //dv_p->scanned_p          = new cub_device_vector<float>(n_element);
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem new scanned_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after scanned vector allocation");

    // Power normalization
    compute_baseline            (dv_p, n_fc, n_element, smooth_scale);     
    if(track_gpu_memory) get_gpu_mem_info("right after baseline computation");
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    //..delete(dv_p->scanned_p);          
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete scanned_p time");

    if(track_gpu_memory) get_gpu_mem_info("right after scanned vector deletion");
    normalize_power_spectrum    (dv_p);

    // Hit finding
    if(track_gpu_memory) get_gpu_mem_info("right after spectrum normalization");
    nhits = find_hits           (dv_p, n_element, maxhits, power_thresh);
    if(track_gpu_memory) get_gpu_mem_info("right after find hits");
    // TODO should probably report if nhits == maxgpuhits, ie overflow
    
    // copy to return vector
    nhits = nhits > maxhits ? maxhits : nhits;
    if(use_timer) timer_start(timer);
    total_nhits += nhits;
    s6_output_block->header.nhits[bors] = nhits;
    // We output both detected and mean powers (not S/N).
    thrust::copy(dv_p->hit_powers_p->begin(),    dv_p->hit_powers_p->end(),    &s6_output_block->power[bors][0]);      
    thrust::copy(dv_p->hit_baselines_p->begin(), dv_p->hit_baselines_p->end(), &s6_output_block->baseline[bors][0]);
    thrust::copy(dv_p->hit_indices_p->begin(),   dv_p->hit_indices_p->end(),   &s6_output_block->hit_indices[bors][0]);
    for(size_t i=0; i<nhits; ++i) {
        long hit_index                        = s6_output_block->hit_indices[bors][i]; 
        long spectrum_index                   = (long)floor((double)hit_index/n_fc);
#ifdef SOURCE_S6
        s6_output_block->pol[bors][i]         = ao_pol(spectrum_index);
        s6_output_block->coarse_chan[bors][i] = ao_coarse_chan(spectrum_index);
#elif SOURCE_DIBAS
        s6_output_block->pol[bors][i]         = dibas_pol(spectrum_index);    
        s6_output_block->coarse_chan[bors][i] = dibas_coarse_chan(spectrum_index, bors);
#elif SOURCE_FAST
        s6_output_block->pol[bors][i]         = pol;   
        s6_output_block->coarse_chan[bors][i] = 0;  // 1 coarse channel for FAST, thus cc number is always 0
#endif
        s6_output_block->fine_chan[bors][i]   = hit_index % n_fc;
//#define PRINT_HIT_INFO
#ifdef PRINT_HIT_INFO
        fprintf(stderr, "bors %d i %d hit_index %ld spectrum_index %ld pol %d cchan %d fchan %d power %f\n", 
                bors, i, hit_index, spectrum_index, s6_output_block->pol[bors][i], s6_output_block->coarse_chan[bors][i], 
                s6_output_block->fine_chan[bors][i], s6_output_block->power[bors][i]);
#endif
    } // end for i<nhits 
    if(use_timer) sum_of_times += timer_stop(timer, "Copy to return vector time");
        
    // delete remaining GPU memory
if(use_thread_sync) hipDeviceSynchronize();

    if(use_mem_timer) timer_start(mem_timer);
    //..delete dv_p->powspec_p;          
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete powspec_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //..delete dv_p->baseline_p;         
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete baseline_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //..delete(dv_p->normalised_p);       
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete nomalised_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //..delete(dv_p->hit_baselines_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_baselines_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //..delete(dv_p->hit_indices_p);  
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_indices_p time");

    if(use_mem_timer) timer_start(mem_timer);
    //..delete(dv_p->hit_powers_p); 
    if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem delete hit_powers_p time");

    //delete(dv_p->raw_timeseries_p);   

//print_current_time("right after sem post");

    //if(use_mem_timer) timer_start(mem_timer);
    //get_singleton_device_allocator()->free_all_cached();    // free all cub allocations
   	//if(use_mem_timer) sum_of_mem_times += timer_stop(mem_timer, "mem free_all_cached 2 time");

	sem_post(gpu_sem);

    if(use_total_gpu_timer) total_gpu_timer.stop();
    if(use_total_gpu_timer) cout << "Sum of GPU times:         \t" << sum_of_times << endl;
    if(use_mem_timer)       cout << "Sum of mem times:         \t" << sum_of_mem_times << endl;    
    if(use_sem_timer)       cout << "Sem time:                 \t" << sem_time << endl;    
    if(use_total_gpu_timer) cout << "Uncounted time:           \t" << total_gpu_timer.getTime() - (sum_of_times + sum_of_mem_times + sem_time) << endl;
    if(use_total_gpu_timer) cout << "Total spectroscopy() time:\t" << total_gpu_timer.getTime() << endl;
    if(use_total_gpu_timer) total_gpu_timer.reset();

    cout<<"------------------------------------------------------------------------------------------"<<endl;
    if(track_gpu_memory) get_gpu_mem_info("right before return to gpu thread");
    return total_nhits;
}

#endif		// REALLOC_x
#endif		// FAST
